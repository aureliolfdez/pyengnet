#include "hip/hip_runtime.h"
#include "main.h"
using namespace std;

__constant__ ulong cols;		  // 8 bytes
__constant__ ulong rows;		  // 8 bytes
__constant__ int maxValueDatasetGPU;		  // 8 bytes
__device__ unsigned long long int numResultKendalls = 0;
__device__ unsigned long long int numResultSpearmans = 0;
__device__ unsigned long long int numResultNMI = 0;

// Kendall
__global__ void kendallTwoGenes(ulong lCombination, double *aResultKendalls, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
	ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
	if (pattern < maxPairs)
	{		
		
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{			
			int iConcordant = 0, iDiscordant = 0, tiersGene1 = 0, tiersGene2 = 0;
			double dKendall = -1;

			// 1) Calc maxValue index of Col1
			int iPosMaxValue = 0;
			float fMaxValue = *(mDataGPU + r1 * cols + 0);
			for (int iCol1 = 1; iCol1 < cols; iCol1++)
			{
				if(fMaxValue < *(mDataGPU + r1 * cols + iCol1)){
					fMaxValue = *(mDataGPU + r1 * cols + iCol1);
					iPosMaxValue = iCol1;
				}
			}

			// 2) Calc concordant and discordant pairs	
			for (int iCol1 = 0; iCol1 < cols; iCol1++)
			{
				if(iCol1 != iPosMaxValue)
				{
					for (int iCol2 = 0; iCol2 < cols; iCol2++)
					{		
																				
						if(iCol1 != iCol2 && *(mDataGPU + r1 * cols + iCol1) < *(mDataGPU + r1 * cols + iCol2))
						{								
							if (*(mDataGPU + r2 * cols + iCol2) > *(mDataGPU + r2 * cols + iCol1))
							{
								iConcordant += 1;
							}
							
							if(*(mDataGPU + r2 * cols + iCol2) < *(mDataGPU + r2 * cols + iCol1))
							{
								iDiscordant += 1;
							}
						}											
					}
				}				
			}

			// 3) Calc tiers
			for (int iCol1 = 0; iCol1 < cols; iCol1++)
			{
				for (int iCol2 = iCol1 + 1; iCol2 < cols; iCol2++)
				{
					// Gene 1
					if(*(mDataGPU + r1 * cols + iCol1) == *(mDataGPU + r1 * cols + iCol2)){
						tiersGene1 += 1;
					}

					if(*(mDataGPU + r2 * cols + iCol1) == *(mDataGPU + r2 * cols + iCol2)){
						tiersGene2 += 1;
					}
				}
			}

			dKendall = (double)(iConcordant - iDiscordant) / (double)(sqrtf((lCombination - tiersGene1) * (lCombination - tiersGene2)));

			*(aResultKendalls + idTh) = dKendall;
		}		
	}
}

// Spearman
/*__global__ void spearmanCalcRanks(double *fSpearmanStats, double *fSpearmanRankG1, double *fSpearmanRankG2, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs)
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			double dMeanG1 = 0, dMeanG2 = 0;
			*(fSpearmanStats + idTh * 4 + 0) = 0;
			*(fSpearmanStats + idTh * 4 + 1) = 0;
			for (int iConditions = 0; iConditions < cols; iConditions++)
			{
				double numEqualOrdG1 = 1.0, readG1 = 1.0, numEqualOrdG2 = 1.0, readG2 = 1.0;
				for (int iCont = 0; iCont < cols; iCont++)
				{
					if (iCont != iConditions)
					{
						if (*(mDataGPU + r1 * cols + iCont) < *(mDataGPU + r1 * cols + iConditions))
						{
							readG1 += 1.0;
						}
						else if (*(mDataGPU + r1 * cols + iCont) == *(mDataGPU + r1 * cols + iConditions))
						{
							numEqualOrdG1 += 1.0;
						}

						if (*(mDataGPU + r2 * cols + iCont) < *(mDataGPU + r2 * cols + iConditions))
						{
							readG2 += 1.0;
						}
						else if (*(mDataGPU + r2 * cols + iCont) == *(mDataGPU + r2 * cols + iConditions))
						{
							numEqualOrdG2 += 1.0;
						}
					}
				}
				*(fSpearmanRankG1 + idTh * cols + iConditions) = readG1 + ((numEqualOrdG1-1.0) * 0.5);
				*(fSpearmanRankG2 + idTh * cols + iConditions) = readG2 + ((numEqualOrdG2-1.0) * 0.5);
				*(fSpearmanStats + idTh * 4 + 0) += *(fSpearmanRankG1 + idTh * cols + iConditions);
				*(fSpearmanStats + idTh * 4 + 1) += *(fSpearmanRankG2 + idTh * cols + iConditions);				
			}

			*(fSpearmanStats + idTh * 4 + 0) = *(fSpearmanStats + idTh * 4 + 0) / cols;
			*(fSpearmanStats + idTh * 4 + 1) = *(fSpearmanStats + idTh * 4 + 1) / cols;
		}
	}
}*/

__global__ void spearmanCalcRanks(double *fSpearmanStats, double *fSpearmanRankG1, double *fSpearmanRankG2, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs)
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			*(fSpearmanStats + idTh * 4 + 0) = 0;
			*(fSpearmanStats + idTh * 4 + 1) = 0;
			for (int iConditions = 0; iConditions < cols; iConditions++)
			{
				double numEqualOrdG1 = 1.0, readG1 = 1.0, numEqualOrdG2 = 1.0, readG2 = 1.0;
				for (int iCont = 0; iCont < cols; iCont++)
				{
					if (iCont != iConditions)
					{
						if (*(mDataGPU + r1 * cols + iCont) < *(mDataGPU + r1 * cols + iConditions))
						{
							readG1 += 1.0;
						}
						else if (*(mDataGPU + r1 * cols + iCont) == *(mDataGPU + r1 * cols + iConditions))
						{
							numEqualOrdG1 += 1.0;
						}

						if (*(mDataGPU + r2 * cols + iCont) < *(mDataGPU + r2 * cols + iConditions))
						{
							readG2 += 1.0;
						}
						else if (*(mDataGPU + r2 * cols + iCont) == *(mDataGPU + r2 * cols + iConditions))
						{
							numEqualOrdG2 += 1.0;
						}
					}
				}
				*(fSpearmanRankG1 + idTh * cols + iConditions) = readG1 + ((numEqualOrdG1-1.0) * 0.5);
				*(fSpearmanRankG2 + idTh * cols + iConditions) = readG2 + ((numEqualOrdG2-1.0) * 0.5);
				*(fSpearmanStats + idTh * 4 + 0) += *(fSpearmanRankG1 + idTh * cols + iConditions);
				*(fSpearmanStats + idTh * 4 + 1) += *(fSpearmanRankG2 + idTh * cols + iConditions);				
			}

			*(fSpearmanStats + idTh * 4 + 0) = *(fSpearmanStats + idTh * 4 + 0) / cols;
			*(fSpearmanStats + idTh * 4 + 1) = *(fSpearmanStats + idTh * 4 + 1) / cols;
		}
	}
}

__global__ void spearmanCovariance(double *fSpearmanStats, double *aResultSpearmans, double *fSpearmanRankG1, double *fSpearmanRankG2, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
    ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
    if (pattern < maxPairs)
	{
		//double dCovariance = 0;
		*(fSpearmanStats + idTh * 4 + 2) = 0;
		for (int iConditions = 0; iConditions < cols; iConditions++)
		{
			*(fSpearmanStats + idTh * 4 + 2) += (*(fSpearmanRankG1 + idTh * cols + iConditions) - *(fSpearmanStats + idTh * 4 + 0)) * (*(fSpearmanRankG2 + idTh * cols + iConditions) - *(fSpearmanStats + idTh * 4 + 1));
		}

		*(aResultSpearmans + idTh) = *(fSpearmanStats + idTh * 4 + 2) / (cols - 1);
	}
}

__global__ void spearmanCalcValue(double *fSpearmanStats, double *aResultSpearmans, double *fSpearmanRankG1, double *fSpearmanRankG2, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
    ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
    if (pattern < maxPairs)
	{
		//double dStdG1 = 0, dStdG2 = 0;
		*(fSpearmanStats + idTh * 4 + 2) = 0;
		*(fSpearmanStats + idTh * 4 + 3) = 0;
		for (int iConditions = 0; iConditions < cols; iConditions++)
		{
			*(fSpearmanStats + idTh * 4 + 2) += (*(fSpearmanRankG1 + idTh * cols + iConditions) - *(fSpearmanStats + idTh * 4 + 0)) * (*(fSpearmanRankG1 + idTh * cols + iConditions) - *(fSpearmanStats + idTh * 4 + 0));
			*(fSpearmanStats + idTh * 4 + 3) += (*(fSpearmanRankG2 + idTh * cols + iConditions) - *(fSpearmanStats + idTh * 4 + 1)) * (*(fSpearmanRankG2 + idTh * cols + iConditions) - *(fSpearmanStats + idTh * 4 + 1));
		}

		*(fSpearmanStats + idTh * 4 + 2) = (double)(sqrtf(*(fSpearmanStats + idTh * 4 + 2) / (cols - 1)));
		*(fSpearmanStats + idTh * 4 + 3) = (double)(sqrtf(*(fSpearmanStats + idTh * 4 + 3) / (cols - 1)));

		*(aResultSpearmans + idTh) = *(aResultSpearmans + idTh) / (*(fSpearmanStats + idTh * 4 + 2) * *(fSpearmanStats + idTh * 4 + 3));
	}
}

// NMI
__global__ void nmiCalcMutualInformation(float *dNMIResults, ulong maxPairs, int id, ulong pairsPerGpuPrevious, int *mDataNormalizedGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor){
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs)
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{

			// Clean dNMIResults by GPU device
			for (int i = 3; i < ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3); i++){
				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) = -1;
			}	
			
			// dNMIResults
			// [0] --> Mutual information and NMI // [1] -->  entropyGen1 // [2] --> entropyGen2 // [3] - ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3)] --> probMaps (calcMI)
			for (int iColumn = 0; iColumn < cols; ++iColumn) {
				int valGen1Column = *(mDataNormalizedGPU + r1 * (cols + 1) + iColumn);				
				int valGen2Column = *(mDataNormalizedGPU + r2 * (cols + 1) + iColumn);

				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (valGen1Column + 3)) = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (valGen1Column + 3)) + 1;
				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (maxValueDatasetGPU + 3 + valGen2Column)) = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (maxValueDatasetGPU + 3 + valGen2Column)) + 1;
				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + ((valGen1Column + maxValueDatasetGPU * valGen2Column) + (maxValueDatasetGPU + maxValueDatasetGPU) + 3)) = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + ((valGen1Column + maxValueDatasetGPU * valGen2Column) + (maxValueDatasetGPU + maxValueDatasetGPU) + 3)) + 1;
			}

			for (int i = 3; i < ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3); i++) {
				if(*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) != -1){
					*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) = (*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) + 1) / cols;
				}				
			}	

			double mi = 0;
			for (int iCont = 0; iCont < (maxValueDatasetGPU * maxValueDatasetGPU); iCont++) {
				if(*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + ((maxValueDatasetGPU + maxValueDatasetGPU) + 3 + iCont)) > 0){
					float doubleValue = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + ((maxValueDatasetGPU + maxValueDatasetGPU) + 3 + iCont));
					float doubleValue2 = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + ((iCont%maxValueDatasetGPU)+3));
					float doubleValue3 = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + ((iCont/maxValueDatasetGPU)+3+maxValueDatasetGPU));
					if(doubleValue > 0 && doubleValue2 > 0 && doubleValue3 > 0){
						mi += doubleValue * logf(doubleValue / doubleValue2 / doubleValue3);
					}					
				}
			}	
			
			mi = mi / logf(2);	
			*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 0) = mi;
		}
	}
}

__global__ void nmiCalcEntropy(float *dNMIResults, ulong maxPairs, int id, ulong pairsPerGpuPrevious, int *mDataNormalizedGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor){
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
    ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
    if (pattern < maxPairs)
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			int maxValGene1 = *(mDataNormalizedGPU + r1 * (cols + 1) + cols);
			int maxValGene2 = *(mDataNormalizedGPU + r2 * (cols + 1) + cols);
			
			// Clean auxiliar data
			for (int i = 3; i < ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3); i++){
				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) = -1;
			}

			// dNMIResults
			// [0] --> Mutual information and NMI // [1] -->  entropyGen1 // [2] --> entropyGen2 // [3 - ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3)] --> probMaps (calcMI)
			for (int iColumn = 0; iColumn < cols; ++iColumn) {
				int valGen1Column = *(mDataNormalizedGPU + r1 * (cols + 1) + iColumn);				
				int valGen2Column = *(mDataNormalizedGPU + r2 * (cols + 1) + iColumn);
				
				// valGen1Column
				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (valGen1Column + 3)) = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (valGen1Column + 3)) + 1;
				
				// valGen2Column
				*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (maxValueDatasetGPU + 3 + valGen2Column)) = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (maxValueDatasetGPU + 3 + valGen2Column)) + 1;
			}

			for (int i = 3; i < ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3); i++) {
				if(*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) != -1){
					*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) = (*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + i) + 1) / cols;
				}				
			}	

			float dEntropyG1 = 0;
			for(int i = 0; i < maxValGene1; i++){
				float varAuxG1 = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (3 + i));
				if(varAuxG1 > 0){
					dEntropyG1 = dEntropyG1 - varAuxG1 * logf(varAuxG1);
				}
			}

			float dEntropyG2 = 0;
			for(int i = 0; i < maxValGene2; i++){				
				float varAuxG2 = *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + (3 + i + maxValueDatasetGPU));
				if(varAuxG2 > 0){
					dEntropyG2 = dEntropyG2 - varAuxG2 * logf(varAuxG2);
				}
			}

			dEntropyG1 = dEntropyG1 / logf(2);		
			dEntropyG2 = dEntropyG2 / logf(2);	
			*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 1) = dEntropyG1;
			*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 2) = dEntropyG2;
		}
	}
}

__global__ void nmiCalcValue(float *dNMIResults, ulong maxPairs, int id, ulong pairsPerGpuPrevious, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor){
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs)
		{
			float dNMI = 0;
			float denom = (*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 1) + *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 2));
			if(denom > 0){
				dNMI = 2.0 * *(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 0) / denom;
			}

			*(dNMIResults + idTh * ((maxValueDatasetGPU + maxValueDatasetGPU) + (maxValueDatasetGPU * maxValueDatasetGPU) + 3) + 0) = dNMI;
		}
}

// #############
// # GPU FUNCS #
// #############
void getNumPairs()
{
	maxPairs = 0;
	for (int i = 0; i < ulRowsData; i++)
	{
		for (int j = i + 1; j < ulRowsData; j++)
		{
			maxPairs++;
		}
	}
}

void prepareGpu1D(ulong lNumber)
{
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	lastBlocksGrid = 1;
	maxIteratorGPU = 0;
	maxThreadsPerBlock = lNumber; // Case 1: 0 < lNumber <= prop.maxThreadsPerBlock
	if (lNumber > prop.maxThreadsPerBlock)
	{ // Case 2: lNumber > prop.maxThreadsPerBlock && Supported GPU in a for
		maxThreadsPerBlock = prop.maxThreadsPerBlock;
		maxBlocksPerGrid = lNumber / prop.maxThreadsPerBlock;
		lastBlocksGrid = lNumber / prop.maxThreadsPerBlock;
		if (lNumber % prop.maxThreadsPerBlock != 0)
		{
			maxBlocksPerGrid++;
			lastBlocksGrid++;
		}
		
		if (maxBlocksPerGrid > prop.maxGridSize[1])
		{ // Case 3: Not supported GPU with a for --> Split patterns in multiple for
			maxIteratorGPU = maxBlocksPerGrid / prop.maxGridSize[1];
			lastBlocksGrid = maxBlocksPerGrid - (maxIteratorGPU * prop.maxGridSize[1]);
			maxBlocksPerGrid = prop.maxGridSize[1];
		}
	}
}

long long *getPairsFiltered(int id, ulong numPatterns)
{
	long long *patFiltered;
	hipMalloc((void **)&patFiltered, numPatterns * sizeof(long long));
	hipMemset(patFiltered, -1, numPatterns * sizeof(long long));
	return patFiltered;
}

void threadsPerDevice(int id, hipStream_t s, ulong chunks,
					  ulong pairsPerGpuPrevious, ulong pairsPerRun, float *mDataGPU, int *mDataNormalizedGPU,
					  mutex *m)
{

	hipSetDevice(id);
	ulong totalPairs = 0;
	ofstream fileResults;
	
	if (bOutput)
	{
		fileResults.open("results/results_GPU_" + to_string(id) + ".csv");
	}
	
	for (ulong largeScale = 0; largeScale < chunks; largeScale++)
	{
		hipSetDevice(id);
		// 1) Validation
		double *aResultKendalls, *aResultSpearmans, *fSpearmanRankG1, *fSpearmanRankG2, *fSpearmanStats;
		float *dNMIResults;
		unsigned long long int numResultKendallsCpu = 0;
		unsigned long long int numResultSpearmanCpu = 0;
		unsigned long long int numResultNMICpu = 0;
		hipMalloc((void **)&aResultKendalls, pairsPerRun * sizeof(double));
		hipMemset(aResultKendalls, 0, pairsPerRun * sizeof(double));
		hipMalloc((void **)&aResultSpearmans, pairsPerRun * sizeof(double));
		hipMemset(aResultSpearmans, 0, pairsPerRun * sizeof(double));
		hipMalloc((void **)&fSpearmanRankG1, pairsPerRun * ulColsData * sizeof(double));
		hipMemset(fSpearmanRankG1, 0, pairsPerRun * ulColsData * sizeof(double));
		hipMalloc((void **)&fSpearmanRankG2, pairsPerRun * ulColsData * sizeof(double));
		hipMemset(fSpearmanRankG2, 0, pairsPerRun * ulColsData * sizeof(double));
		hipMalloc((void **)&fSpearmanStats, pairsPerRun * 4 * sizeof(double));
		hipMemset(fSpearmanStats, 0, pairsPerRun * 4 * sizeof(double));
		hipMalloc((void **)&dNMIResults, pairsPerRun * ((maxValueDataset + maxValueDataset) + (maxValueDataset * maxValueDataset) + 3) * sizeof(float));
		hipMemset(dNMIResults, 0, pairsPerRun * ((maxValueDataset + maxValueDataset) + (maxValueDataset * maxValueDataset) + 3) * sizeof(float));

		// Kendall
		ulong lCombination = 0;
		for(ulong i=1; i < ulColsData; i++){
			lCombination += i;
		}
		prepareGpu1D(pairsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			kendallTwoGenes<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(lCombination, aResultKendalls, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		kendallTwoGenes<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(lCombination, aResultKendalls, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		// Spearman
		prepareGpu1D(pairsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			spearmanCalcRanks<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(fSpearmanStats, fSpearmanRankG1, fSpearmanRankG2, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		spearmanCalcRanks<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(fSpearmanStats, fSpearmanRankG1, fSpearmanRankG2, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			spearmanCovariance<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(fSpearmanStats, aResultSpearmans, fSpearmanRankG1, fSpearmanRankG2, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		spearmanCovariance<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(fSpearmanStats, aResultSpearmans, fSpearmanRankG1, fSpearmanRankG2, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			spearmanCalcValue<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(fSpearmanStats, aResultSpearmans, fSpearmanRankG1, fSpearmanRankG2, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		spearmanCalcValue<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(fSpearmanStats, aResultSpearmans, fSpearmanRankG1, fSpearmanRankG2, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);
		
		// NMI
		prepareGpu1D(pairsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			nmiCalcMutualInformation<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		nmiCalcMutualInformation<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			nmiCalcEntropy<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		nmiCalcEntropy<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			nmiCalcValue<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		nmiCalcValue<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		// 2) Save results
		if (bOutput)
		{

			// Transfer global memory to RAM
			double *aResultKendallsCpu = (double *)malloc(pairsPerRun * sizeof(double));
			hipMemcpy(aResultKendallsCpu, aResultKendalls, pairsPerRun * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpyFromSymbol(&numResultKendallsCpu, HIP_SYMBOL(numResultKendalls),
								 sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

			double *aResultSpearmansCpu = (double *)malloc(pairsPerRun * sizeof(double));
			hipMemcpy(aResultSpearmansCpu, aResultSpearmans, pairsPerRun * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpyFromSymbol(&numResultKendallsCpu, HIP_SYMBOL(numResultKendalls),
								 sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
			
			float *dNMIResultsCpu = (float *)malloc(pairsPerRun * ((maxValueDataset + maxValueDataset) + (maxValueDataset * maxValueDataset) + 3) * sizeof(float));
			hipMemcpy(dNMIResultsCpu, dNMIResults, pairsPerRun * ((maxValueDataset + maxValueDataset) + (maxValueDataset * maxValueDataset) + 3) * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpyFromSymbol(&numResultNMICpu, HIP_SYMBOL(numResultNMI),
								 sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

			if(largeScale == 0){
				fileResults << "g1;g2;nmi;kendall;spearman"
						<< "\n";
			}
			for (ulong iRow = 0; iRow < pairsPerRun; iRow++)
			{
				ulong pattern = iRow + pairsPerGpuPrevious;

				if (pattern < maxPairs)
				{
					
					long r1 = 0;
					long r2 = -1;
					long auxPat = pattern - ulRowsData + 1;
					if (auxPat < 0)
					{
						r2 = auxPat + ulRowsData;
					}
					for (ulong j = ulRowsData - 2; r2 == -1; j--)
					{
						auxPat = auxPat - j;
						r1++;
						if (auxPat < 0)
						{
							r2 = (j + auxPat) + (r1 + 1);
						}
					}

					int iMajorVoting = 0;
					double dKendall = *(aResultKendallsCpu + iRow);
					double dSpearman = *(aResultSpearmansCpu + iRow);
					float dNMI = *(dNMIResultsCpu + iRow * ((maxValueDataset + maxValueDataset) + (maxValueDataset * maxValueDataset) + 3) + 0);

					if(dKendall < 0){
						dKendall = dKendall * -1;
					}

					if(dSpearman < 0){
						dSpearman = dSpearman * -1;
					}

					if(dNMI < 0){
						dNMI = dNMI * -1;
					}

					if(dKendall >= correctionThresholdKendall){
						iMajorVoting++;
					}
	
					if(dSpearman >= correctionThresholdSpearman){
						iMajorVoting++;
					}

					if(dNMI >= correctionThresholdNMI){
						iMajorVoting++;
					}
					
					if (iMajorVoting >= 2)
					{
						fileResults << geneNames[r1] << ";" << geneNames[r2] << ";" << dNMI << ";" << dKendall << ";" << dSpearman << "\n";
					}
				}
			}

			free(aResultKendallsCpu);
			free(aResultSpearmansCpu);
			free(dNMIResultsCpu);
		}

		hipFree(aResultKendalls);
		hipFree(aResultSpearmans);
		hipFree(fSpearmanRankG1);
		hipFree(fSpearmanRankG2);
		hipFree(fSpearmanStats);
		hipFree(dNMIResults);
		numResultKendallsCpu = 0;
		numResultSpearmanCpu = 0;
		numResultNMICpu = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(numResultKendalls), &numResultKendallsCpu, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(numResultSpearmans), &numResultSpearmanCpu, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);	
		hipMemcpyToSymbol(HIP_SYMBOL(numResultNMI), &numResultNMICpu, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);			
		totalPairs += pairsPerRun;
	}

	if (bOutput)
	{
		fileResults.close();
	}
}

double runAlgorithm()
{
	for (int i = 0; i < iDeviceCount; ++i)
	{
		hipSetDevice(i);
		hipMemcpyToSymbol(*(&cols), &ulColsData, sizeof(ulong), 0,
						   hipMemcpyHostToDevice);
		hipMemcpyToSymbol(*(&rows), &ulRowsData, sizeof(ulong), 0,
						   hipMemcpyHostToDevice);
		hipMemcpyToSymbol(*(&maxValueDatasetGPU), &maxValueDataset, sizeof(int), 0,
						   hipMemcpyHostToDevice);
	}

	getNumPairs();

	// 2) PREPARING LARGE-SCALE DATA: CHUNKS
	hipStream_t s[iDeviceCount];
	thread threads[iDeviceCount];
	ulong chunks[iDeviceCount], pairsPerRun[iDeviceCount];
	ulong pairsPerGpu = maxPairs / iDeviceCount;
	ulong restPairsPerGpu = maxPairs % iDeviceCount;

	for (int i = 0; i < iDeviceCount; ++i)
	{
		hipSetDevice(i);
		hipStreamCreate(&s[i]);
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		double availableMemory = ((3 * prop.totalGlobalMem) / 4 - (ulRowsData * ulColsData * sizeof(float)) - (ulRowsData * (ulColsData+1) * sizeof(int))); // mData + mDataNormalized
		double sizeResultKendall = 0, sizeResultsSpearman = 0, fSpearmanRankG1 = 0, fSpearmanRankG2 = 0, fSpearmanStats = 0;
		float dNMIResults = 0;
		sizeResultKendall = (pairsPerGpu * sizeof(double));
		sizeResultsSpearman = (pairsPerGpu * sizeof(double));
		fSpearmanRankG1 = (pairsPerGpu * ulColsData * sizeof(double));
		fSpearmanRankG2 = (pairsPerGpu * ulColsData * sizeof(double));
		fSpearmanStats = (pairsPerGpu * 4 * sizeof(double)); // [1] --> Mean of rank G1; [2] --> Mean of rank G2; ; [3] --> aux ; [4] --> aux
		dNMIResults = (pairsPerGpu * ((maxValueDataset + maxValueDataset) + (maxValueDataset * maxValueDataset) + 3) * sizeof(float)); // [0] --> Mutual information ::: NMI [1] --> entropyGen1 ::: [2] --> entropyGen2 ::: [3 - maxValueDataset] --> probMaps
		chunks[i] = ((sizeResultKendall + sizeResultsSpearman + fSpearmanRankG1 + fSpearmanRankG2 + fSpearmanStats + dNMIResults) / availableMemory) + 1;
		pairsPerRun[i] = pairsPerGpu / chunks[i];

		if (pairsPerGpu % chunks[i] != 0)
		{
			pairsPerRun[i]++;
		}
		if (iDeviceCount > 1 && maxPairs % iDeviceCount != 0 && i == iDeviceCount - 1)
		{
			pairsPerRun[i] += restPairsPerGpu;
		}
	}

	ulong pairsPerGpuPrevious = 0;

	mutex m;
	struct timeval stop, start;
	float *mDataGPU;
	int *mDataNormalizedGPU;
	for (int i = 0; i < iDeviceCount; ++i)
	{
		hipSetDevice(i);
		hipHostMalloc((void **)&mDataGPU, ulRowsData * ulColsData * sizeof(float));
		hipMemcpy(mDataGPU, mData, ulRowsData * ulColsData * sizeof(float),
				   hipMemcpyHostToDevice);
		hipHostMalloc((void **)&mDataNormalizedGPU, ulRowsData * (ulColsData+1) * sizeof(int));
		hipMemcpy(mDataNormalizedGPU, mDataNormalized, ulRowsData * (ulColsData+1) * sizeof(int),
							  hipMemcpyHostToDevice); 
	}
	for (int i=0; i < iDeviceCount; ++i)
	{
		if (i > 0)
		{
			pairsPerGpuPrevious += chunks[i - 1] * pairsPerRun[i - 1];
		}

		threads[i] = thread(threadsPerDevice, i, s[i], chunks[i], pairsPerGpuPrevious, pairsPerRun[i], mDataGPU, mDataNormalizedGPU, &m);
	}

	gettimeofday(&start, NULL);
	for (auto &th : threads)
	{
		th.join();
	}
	gettimeofday(&stop, NULL);


	free(mData);
	for (int i = 0; i < iDeviceCount; ++i)
    {
        hipSetDevice(i);
        hipDeviceReset();
    }
	return (((stop.tv_sec - start.tv_sec) * 1000.0) + ((stop.tv_usec - start.tv_usec) / 1000.0)) / 1000.0;
}

