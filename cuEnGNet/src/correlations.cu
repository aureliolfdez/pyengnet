#include "hip/hip_runtime.h"
#include "main.h"
#include <thread>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <stdint.h>
#include <sys/time.h>
#include <unistd.h>
#include <inttypes.h>
#include <iterator>
#include <utility>
#include <algorithm>
#include <string>
#include <sstream>
#include <fstream>
#include <cstdlib>
#include <time.h>
#include <set>
#include <vector>
#include <map>
#include <unordered_set>
#include <mutex>
using namespace std;

__constant__ ulong cols;		  // 8 bytes
__constant__ ulong rows;		  // 8 bytes
__device__ unsigned long long int numResultKendalls = 0;
__device__ unsigned long long int numResultSpearmans = 0;
__device__ unsigned long long int numResultNMI = 0;

__global__ void kendallTwoGenes(double *aResultKendalls, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor, float correctionThreshold)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
	ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
	if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{		
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{			
			int iConcordant = 0, iDiscordant = 0;
			double dKendall = -1;

			for (int iCol1 = 0; iCol1 < cols; iCol1++)
			{
				float fValueR1 = *(mDataGPU + r1 * cols + iCol1);
				float fValueR2 = *(mDataGPU + r2 * cols + iCol1);
				
				for (int iCol2 = 0; iCol2 < cols; iCol2++)
				{
					if (*(mDataGPU + r1 * cols + iCol2) > fValueR1)
					{
						if (*(mDataGPU + r2 * cols + iCol2) > fValueR2)
						{
							iConcordant += 1;
						}
						else
						{
							iDiscordant += 1;
						}
					}
				}
			}

			if (iConcordant + iDiscordant != 0)
			{ // Control division by zero
				dKendall = (double)(iConcordant - iDiscordant) / (iConcordant + iDiscordant);
				if (dKendall < 0)
				{ // Absolute value
					dKendall = dKendall * -1;
				}
				if (dKendall <= correctionThreshold)
				{
					dKendall = -1;
				}
			}

			*(aResultKendalls + idTh) = dKendall;
		}
	}
}

__global__ void spearmanCalcfDiGenesOne(double *fDiSpearman, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			for (int iConditions = 0; iConditions < cols; iConditions++)
			{
				int numEqualOrdX = 0;
				for (int iCont = 0; iCont < cols; iCont++)
				{
					if (iCont != iConditions)
					{
						if (*(mDataGPU + r1 * cols + iCont) < *(mDataGPU + r1 * cols + iConditions))
						{
							*(fDiSpearman + idTh * cols + iConditions) += 1;
						}
						else if (*(mDataGPU + r1 * cols + iCont) == *(mDataGPU + r1 * cols + iConditions))
						{
							numEqualOrdX += 1;
						}
					}
				}

				if (numEqualOrdX == 0)
				{
					*(fDiSpearman + idTh * cols + iConditions) = *(fDiSpearman + idTh * cols + iConditions) + 1;
				}
				else
				{
					*(fDiSpearman + idTh * cols + iConditions) = *(fDiSpearman + idTh * cols + iConditions) + 1 + (numEqualOrdX / 2.0);
				}
			}
		}
	}
}

__global__ void spearmanCalcfDiGenesTwo(double *fDiSpearman, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			for (int iConditions = 0; iConditions < cols; iConditions++)
			{
				int numEqualOrdY = 0;
				float fValueR2 = *(mDataGPU + r2 * cols + iConditions);
				double dFi = *(fDiSpearman + idTh * cols + iConditions);
				for (int iCont = 0; iCont < cols; iCont++)
				{
					if (iCont != iConditions)
					{
						float fValueCont = *(mDataGPU + r2 * cols + iCont);
						if (fValueCont < fValueR2)
						{
							dFi -= 1;
						}
						else if (fValueCont == fValueR2)
						{
							numEqualOrdY += 1;
						}
					}
				}
				
				if (numEqualOrdY == 0)
				{
					dFi = dFi - 1;
				}
				else
				{
					dFi = dFi - 1 - (numEqualOrdY / 2.0);
				}

				*(fDiSpearman + idTh * cols + iConditions) = dFi;
			}
		}
	}
}

__global__ void spearmanCalc(double *aResultSpearmans, double *fDiSpearman, ulong maxPairs, int id, ulong pairsPerGpuPrevious, float *mDataGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor, float correctionThreshold)
{
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{
			double diSquare = 0;
			for (int iConditions = 0; iConditions < cols; iConditions++)
			{
				diSquare = diSquare + (*(fDiSpearman + idTh * cols + iConditions) * *(fDiSpearman + idTh * cols + iConditions));
			}
			double dSpearman = 1 - ((6 * diSquare) / (cols * ((cols * cols) - 1)));
			if(dSpearman < 0){
				dSpearman = dSpearman * -1;
			}
			if(dSpearman <= correctionThreshold){
				dSpearman = -1;
			}
			*(aResultSpearmans + idTh) = dSpearman;
	}
}

__global__ void nmiCalcMutualInformation(float *dNMIResults, ulong maxPairs, int id, ulong pairsPerGpuPrevious, int *mDataNormalizedGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor){
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			int maxVal = *(mDataNormalizedGPU + r1 * (cols + 1) + cols);

			// Clean dNMIResults by GPU device
			for (int iColumn = 0; iColumn < 11; ++iColumn) {
				*(dNMIResults + idTh * 11 + iColumn) = 0;
			}	
			
			// dNMIResults
			// [0] --> Mutual information and NMI // [1] -->  entropyGen1 // [2] --> entropyGen2 // [3 - 10] --> probMaps (calcMI)
			for (int iColumn = 0; iColumn < cols; ++iColumn) {
				int valGen1Column = *(mDataNormalizedGPU + r1 * (cols + 1) + iColumn);
				int valGen2Column = *(mDataNormalizedGPU + r2 * (cols + 1) + iColumn);	

				*(dNMIResults + idTh * 11 + (valGen1Column + 3)) = *(dNMIResults + idTh * 11 + (valGen1Column + 3)) + 1;
				*(dNMIResults + idTh * 11 + (valGen2Column + 5)) = *(dNMIResults + idTh * 11 + (valGen2Column + 5)) + 1;
				*(dNMIResults + idTh * 11 + ((valGen1Column + maxVal * valGen2Column) + 7)) = *(dNMIResults + idTh * 11 + ((valGen1Column + maxVal * valGen2Column) + 7)) + 1;
			}

			for (int iCont = 0; iCont < 8; iCont++) {
				*(dNMIResults + idTh * 11 + (3 + iCont)) = *(dNMIResults + idTh * 11 + (3 + iCont)) / cols;
			}			

			double mi = 0;
			for (int iCont = 0; iCont < 4; iCont++) {
				if (*(dNMIResults + idTh * 11 + (7 + iCont)) > 0 && *(dNMIResults + idTh * 11 + ((iCont%maxVal)+3)) > 0 && *(dNMIResults + idTh * 11 + (iCont/maxVal)+5) > 0) {
					mi += *(dNMIResults + idTh * 11 + (7 + iCont)) * logf(*(dNMIResults + idTh * 11 + (7 + iCont)) / *(dNMIResults + idTh * 11 + ((iCont%maxVal)+3)) / *(dNMIResults + idTh * 11 + (iCont/maxVal)+5));
				}
			}		

			mi = mi / logf(2);			
			*(dNMIResults + idTh * 11 + 0) = mi;
		}
	}
}

__global__ void nmiCalcEntropy(float *dNMIResults, ulong maxPairs, int id, ulong pairsPerGpuPrevious, int *mDataNormalizedGPU, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor){
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0)
		{
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--)
		{
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0)
			{
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows)
		{
			// Clean auxiliar data
			*(dNMIResults + idTh * 11 + 3) = 0;
			*(dNMIResults + idTh * 11 + 4) = 0;
			*(dNMIResults + idTh * 11 + 5) = 0;
			*(dNMIResults + idTh * 11 + 6) = 0;
			*(dNMIResults + idTh * 11 + 7) = 0;
			*(dNMIResults + idTh * 11 + 8) = 0;
			*(dNMIResults + idTh * 11 + 9) = 0;
			*(dNMIResults + idTh * 11 + 10) = 0;

			// dNMIResults
			// [0] --> Mutual information and NMI // [1] -->  entropyGen1 // [2] --> entropyGen2 // [3 - 10] --> probMaps (calcMI)
			for (int iColumn = 0; iColumn < cols; ++iColumn) {
				int valGen1Column = *(mDataNormalizedGPU + r1 * (cols + 1) + iColumn);
				int valGen2Column = *(mDataNormalizedGPU + r2 * (cols + 1) + iColumn);

				*(dNMIResults + idTh * 11 + (valGen1Column + 3)) = *(dNMIResults + idTh * 11 + (valGen1Column + 3)) + 1;
				*(dNMIResults + idTh * 11 + (valGen2Column + 5)) = *(dNMIResults + idTh * 11 + (valGen2Column + 5)) + 1;
			}

			for (int iCont = 0; iCont < 4; iCont++) {
				*(dNMIResults + idTh * 11 + (3 + iCont)) = *(dNMIResults + idTh * 11 + (3 + iCont)) / cols;
			}			

			float dEntropyG1, dEntropyG2 = 0;
			for (int iCont = 0; iCont < 2; iCont++) {
				float varAuxG1 = *(dNMIResults + idTh * 11 + (3 + iCont));
				float varAuxG2 = *(dNMIResults + idTh * 11 + (5 + iCont));
				if(varAuxG1 > 0){
					dEntropyG1 = dEntropyG1 - varAuxG1 * logf(varAuxG1);
				}

				if(varAuxG2 > 0){
					dEntropyG2 = dEntropyG2 - varAuxG2 * logf(varAuxG2);
				}
			}		

			dEntropyG1 = dEntropyG1 / logf(2);		
			dEntropyG2 = dEntropyG2 / logf(2);	
			*(dNMIResults + idTh * 11 + 1) = dEntropyG1;
			*(dNMIResults + idTh * 11 + 2) = dEntropyG2;
		}
	}
}

__global__ void nmiCalcValue(float *dNMIResults, ulong maxPairs, int id, ulong pairsPerGpuPrevious, ulong totalPairs, ulong pairsPerRun, int iter, ulong totalFor, float correctionThreshold){
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
        ulong pattern = idTh + (totalFor * (iter - 1)) + pairsPerGpuPrevious + totalPairs;
        if (pattern < maxPairs && pattern < (pairsPerGpuPrevious+totalFor))
	{
		float dNMI = -1;
		float denom = (*(dNMIResults + idTh * 11 + 1) + *(dNMIResults + idTh * 11 + 2));
		if(denom != 0){
			dNMI = 2.0 * *(dNMIResults + idTh * 11 + 0) / (*(dNMIResults + idTh * 11 + 1) + *(dNMIResults + idTh * 11 + 2));
		}

		if(dNMI > 300){
			dNMI = -1;
		}
		
		if(dNMI < correctionThreshold){
			dNMI = -1;
		}

		*(dNMIResults + idTh * 11 + 0) = dNMI;
	}
}

// #############
// # GPU FUNCS #
// #############
void getNumPairs()
{
	maxPairs = 0;
	for (int i = 0; i < ulRowsData; i++)
	{
		for (int j = i + 1; j < ulRowsData; j++)
		{
			maxPairs++;
		}
	}
}

void prepareGpu1D(ulong lNumber)
{
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	lastBlocksGrid = 1;
	maxIteratorGPU = 0;
	maxThreadsPerBlock = lNumber; // Case 1: 0 < lNumber <= prop.maxThreadsPerBlock
	if (lNumber > prop.maxThreadsPerBlock)
	{ // Case 2: lNumber > prop.maxThreadsPerBlock && Supported GPU in a for
		maxThreadsPerBlock = prop.maxThreadsPerBlock;
		maxBlocksPerGrid = lNumber / prop.maxThreadsPerBlock;
		lastBlocksGrid = lNumber / prop.maxThreadsPerBlock;
		if (lNumber % prop.maxThreadsPerBlock != 0)
		{
			maxBlocksPerGrid++;
			lastBlocksGrid++;
		}
		
		if (maxBlocksPerGrid > prop.maxGridSize[1])
		{ // Case 3: Not supported GPU with a for --> Split patterns in multiple for
			maxIteratorGPU = maxBlocksPerGrid / prop.maxGridSize[1];
			lastBlocksGrid = maxBlocksPerGrid - (maxIteratorGPU * prop.maxGridSize[1]);
			maxBlocksPerGrid = prop.maxGridSize[1];
		}
	}
}

long long *getPairsFiltered(int id, ulong numPatterns)
{
	long long *patFiltered;
	hipMalloc((void **)&patFiltered, numPatterns * sizeof(long long));
	hipMemset(patFiltered, -1, numPatterns * sizeof(long long));
	return patFiltered;
}

void threadsPerDevice(int id, hipStream_t s, ulong chunks,
					  ulong pairsPerGpuPrevious, ulong pairsPerRun, float *mDataGPU, int *mDataNormalizedGPU,
					  mutex *m)
{

	hipSetDevice(id);
	ulong totalPairs = 0;
	ofstream fileResults;
	
	if (bOutput)
	{
		fileResults.open("results/results_GPU_" + to_string(id) + ".csv");
	}
	
	for (ulong largeScale = 0; largeScale < chunks; largeScale++)
	{
		hipSetDevice(id);
		// 1) Validation
		double *aResultKendalls, *aResultSpearmans, *fDiSpearman;
		float *dNMIResults;
		unsigned long long int numResultKendallsCpu = 0;
		unsigned long long int numResultSpearmanCpu = 0;
		unsigned long long int numResultNMICpu = 0;
		hipMalloc((void **)&aResultKendalls, pairsPerRun * sizeof(double));
		hipMemset(aResultKendalls, 0, pairsPerRun * sizeof(double));
		hipMalloc((void **)&aResultSpearmans, pairsPerRun * sizeof(double));
		hipMemset(aResultSpearmans, 0, pairsPerRun * sizeof(double));
		hipMalloc((void **)&fDiSpearman, pairsPerRun * ulColsData * sizeof(double));
		hipMemset(fDiSpearman, 0, pairsPerRun * ulColsData * sizeof(double));
		hipMalloc((void **)&dNMIResults, pairsPerRun * 11 * sizeof(float));
		hipMemset(dNMIResults, 0, pairsPerRun * 11 * sizeof(float));

		// Kendall
		prepareGpu1D(pairsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			kendallTwoGenes<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(aResultKendalls, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid, correctionThresholdKendall);
		}
		kendallTwoGenes<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(aResultKendalls, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid, correctionThresholdKendall);

		// Spearman
		prepareGpu1D(pairsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			spearmanCalcfDiGenesOne<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(fDiSpearman, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		spearmanCalcfDiGenesOne<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(fDiSpearman, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			spearmanCalcfDiGenesTwo<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(fDiSpearman, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		spearmanCalcfDiGenesTwo<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(fDiSpearman, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			spearmanCalc<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(aResultSpearmans, fDiSpearman, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid, correctionThresholdSpearman);
		}
		spearmanCalc<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(aResultSpearmans, fDiSpearman, maxPairs, id, pairsPerGpuPrevious, mDataGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid, correctionThresholdSpearman);
		
		// NMI
		prepareGpu1D(pairsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			nmiCalcMutualInformation<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		nmiCalcMutualInformation<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			nmiCalcEntropy<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid);
		}
		nmiCalcEntropy<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, mDataNormalizedGPU, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid);

		for (int i = 1; i <= maxIteratorGPU; i++)
		{
			nmiCalcValue<<<maxBlocksPerGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, totalPairs, pairsPerRun, i, maxThreadsPerBlock * maxBlocksPerGrid, correctionThresholdNMI);
		}
		nmiCalcValue<<<lastBlocksGrid, maxThreadsPerBlock, 0, s>>>(dNMIResults, maxPairs, id, pairsPerGpuPrevious, totalPairs, pairsPerRun, maxIteratorGPU + 1, maxThreadsPerBlock * maxBlocksPerGrid, correctionThresholdNMI);

		// 2) Save results
		if (bOutput)
		{
			// Transfer global memory to RAM
			double *aResultKendallsCpu = (double *)malloc(pairsPerRun * sizeof(double));
			hipMemcpy(aResultKendallsCpu, aResultKendalls, pairsPerRun * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpyFromSymbol(&numResultKendallsCpu, HIP_SYMBOL(numResultKendalls),
								 sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

			double *aResultSpearmansCpu = (double *)malloc(pairsPerRun * sizeof(double));
			hipMemcpy(aResultSpearmansCpu, aResultSpearmans, pairsPerRun * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpyFromSymbol(&numResultKendallsCpu, HIP_SYMBOL(numResultKendalls),
								 sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

			float *dNMIResultsCpu = (float *)malloc(pairsPerRun * 11 * sizeof(float));
			hipMemcpy(dNMIResultsCpu, dNMIResults, pairsPerRun * 11 * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpyFromSymbol(&numResultNMICpu, HIP_SYMBOL(numResultNMI),
								 sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

			fileResults << "g1;g2;kendall_tau;spearman;nmi"
						<< "\n";
			for (ulong iRow = 0; iRow < pairsPerRun; iRow++)
			{
				ulong pattern = iRow + pairsPerGpuPrevious;
				if (pattern < maxPairs && pattern < (pairsPerGpuPrevious + (maxThreadsPerBlock*maxBlocksPerGrid)))
				{
					long r1 = 0;
					long r2 = -1;
					long auxPat = pattern - ulRowsData + 1;
					if (auxPat < 0)
					{
						r2 = auxPat + ulRowsData;
					}
					for (ulong j = ulRowsData - 2; r2 == -1; j--)
					{
						auxPat = auxPat - j;
						r1++;
						if (auxPat < 0)
						{
							r2 = (j + auxPat) + (r1 + 1);
						}
					}

					int iMajorVoting = 0;
					double dKendall = *(aResultKendallsCpu + iRow);
					double dSpearman = *(aResultSpearmansCpu + iRow);
					float dNMI = *(dNMIResultsCpu + iRow * 11 + 0);
					if(dKendall != -1){
						iMajorVoting++;
					}
	
					if(dSpearman != -1){
						iMajorVoting++;
					}

					if(dNMI != -1){
						iMajorVoting++;
					}
					
					if (iMajorVoting >= 2)
					{
						fileResults << r1 << ";" << r2 << ";" << dKendall << ";" << dSpearman << ";" << dNMI << "\n";
					}
				}
			}

			free(aResultKendallsCpu);
			free(aResultSpearmansCpu);
			free(dNMIResultsCpu);
		}

		hipFree(aResultKendalls);
		hipFree(aResultSpearmans);
		hipFree(fDiSpearman);
		hipFree(dNMIResults);
		numResultKendallsCpu = 0;
		numResultSpearmanCpu = 0;
		numResultNMICpu = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(numResultKendalls), &numResultKendallsCpu, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(numResultSpearmans), &numResultSpearmanCpu, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);	
		hipMemcpyToSymbol(HIP_SYMBOL(numResultNMI), &numResultNMICpu, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);			
		totalPairs += pairsPerRun;
	}

	if (bOutput)
	{
		fileResults.close();
	}
}

double runAlgorithm()
{
	for (int i = 0; i < iDeviceCount; ++i)
	{
		hipSetDevice(i);
		hipMemcpyToSymbol(*(&cols), &ulColsData, sizeof(ulong), 0,
						   hipMemcpyHostToDevice);
		hipMemcpyToSymbol(*(&rows), &ulRowsData, sizeof(ulong), 0,
						   hipMemcpyHostToDevice);
	}

	getNumPairs();

	// 2) PREPARING LARGE-SCALE DATA: CHUNKS
	hipStream_t s[iDeviceCount];
	thread threads[iDeviceCount];
	ulong chunks[iDeviceCount], pairsPerRun[iDeviceCount];
	ulong pairsPerGpu = maxPairs / iDeviceCount;
	ulong restPairsPerGpu = maxPairs % iDeviceCount;

	for (int i = 0; i < iDeviceCount; ++i)
	{
		hipSetDevice(i);
		hipStreamCreate(&s[i]);
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		double availableMemory = ((3 * prop.totalGlobalMem) / 4 - (ulRowsData * ulColsData * sizeof(float)) - (ulRowsData * (ulColsData+1) * sizeof(int))); // mData + mDataNormalized
		double sizeResultKendall = 0, sizeResultsSpearman = 0, fDiSpearman = 0;
		float dNMIResults = 0;
		sizeResultKendall = (pairsPerGpu * sizeof(double));
		sizeResultsSpearman = (pairsPerGpu * sizeof(double));
		fDiSpearman = (pairsPerGpu * ulColsData * sizeof(double));
		dNMIResults = (pairsPerGpu * 11 * sizeof(float)); // [0] --> Mutual information and NMI [1] --> entropyGen1 [2] --> entropyGen2 [3 - 10] --> probMaps (calcMI) [3 - 4] --> calcEnropy
		chunks[i] = ((sizeResultKendall + sizeResultsSpearman + fDiSpearman + dNMIResults) / availableMemory) + 1;
		pairsPerRun[i] = pairsPerGpu / chunks[i];

		if (pairsPerGpu % chunks[i] != 0)
		{
			pairsPerRun[i]++;
		}
		if (iDeviceCount > 1 && maxPairs % iDeviceCount != 0 && i == iDeviceCount - 1)
		{
			pairsPerRun[i] += restPairsPerGpu;
		}
	}

	ulong pairsPerGpuPrevious = 0;

	mutex m;
	struct timeval stop, start;
	float *mDataGPU;
	int *mDataNormalizedGPU;
	for (int i = 0; i < iDeviceCount; ++i)
	{
		hipSetDevice(i);
		hipHostMalloc((void **)&mDataGPU, ulRowsData * ulColsData * sizeof(float));
		hipMemcpy(mDataGPU, mData, ulRowsData * ulColsData * sizeof(float),
				   hipMemcpyHostToDevice);
		hipHostMalloc((void **)&mDataNormalizedGPU, ulRowsData * (ulColsData+1) * sizeof(int));
		hipMemcpy(mDataNormalizedGPU, mDataNormalized, ulRowsData * (ulColsData+1) * sizeof(int),
							  hipMemcpyHostToDevice);
	}
	for (int i=0; i < iDeviceCount; ++i)
	{
		if (i > 0)
		{
			pairsPerGpuPrevious += chunks[i - 1] * pairsPerRun[i - 1];
		}

		threads[i] = thread(threadsPerDevice, i, s[i], chunks[i], pairsPerGpuPrevious, pairsPerRun[i], mDataGPU, mDataNormalizedGPU, &m);
	}

	gettimeofday(&start, NULL);
	for (auto &th : threads)
	{
		th.join();
	}
	gettimeofday(&stop, NULL);


	free(mData);
	for (int i = 0; i < iDeviceCount; ++i)
    {
        hipSetDevice(i);
        hipDeviceReset();
    }
	return (((stop.tv_sec - start.tv_sec) * 1000.0) + ((stop.tv_usec - start.tv_usec) / 1000.0)) / 1000.0;
}
